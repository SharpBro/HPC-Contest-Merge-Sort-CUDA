#include "hip/hip_runtime.h"
#include "main.hpp"

__device__ void merge_gpu_streams(DATATYPE *list, DATATYPE *sorted, int start, int mid, int end);
__global__ void mergesort_gpu_streams(DATATYPE *list, DATATYPE *sorted, int n, int chunk);

// // // // // // // // // // // // // // // //
//  GPU Implementation                       //
// // // // // // // // // // // // // // // //
__device__ void merge_gpu_streams(DATATYPE *list, DATATYPE *sorted, int start, int mid, int end) {
    int k = start, i = start, j = mid;
    while (i < mid || j < end)
    {
        if (j == end)
            sorted[k] = list[i++];
        else if (i == mid)
            sorted[k] = list[j++];
        else if (list[i] < list[j])
            sorted[k] = list[i++];
        else
            sorted[k] = list[j++];
        k++;
    }
}

__global__ void mergesort_gpu_streams(DATATYPE *list, DATATYPE *sorted, int n, int chunk) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int start = tid * chunk;
    if (start >= n)
        return;
    int mid, end;

    mid = min(start + chunk / 2, n);
    end = min(start + chunk, n);
    merge_gpu_streams(list, sorted, start, mid, end);
}

int mergesort_streams(DATATYPE *list, DATATYPE *sorted, int n) {

    DATATYPE *list_d;
    DATATYPE *sorted_d;
    int dummy;
    bool flag = false;
    bool sequential = false;

    int size = n * sizeof(DATATYPE);

    hipMalloc((void **)&list_d, size);
    hipMalloc((void **)&sorted_d, size);

    hipStream_t str1,str2,str3;
    hipStreamCreate(&str1);
    hipStreamCreate(&str2);
    hipStreamCreate(&str3);

    hipMemcpyAsync(list_d, list, size, hipMemcpyHostToDevice,str1);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error_2: %s\n", hipGetErrorString(err));
        return -1;
    }

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    // vaues for sm_35 compute capability
    int max_active_blocks_per_sm = 16;
    if(prop.major > 3 && (prop.major < 8 && prop.minor < 5) || prop.major == 8)
        max_active_blocks_per_sm = 32;

    //const int max_active_warps_per_sm = 64;

    int warp_size = prop.warpSize;
    int max_grid_size = prop.maxGridSize[0];
    int max_threads_per_block = prop.maxThreadsPerBlock;
    int max_procs_count = prop.multiProcessorCount;

    int max_active_blocks = max_active_blocks_per_sm * max_procs_count;
    //int max_active_warps = max_active_warps_per_sm * max_procs_count;

    int chunk_size;
    float total_elapsed_time = 0;

    for (chunk_size = 2; chunk_size < 2 * n; chunk_size *= 2) {
        int blocks_required = 0, threads_per_block = 0;
        int threads_required = (n % chunk_size == 0) ? n / chunk_size : n / chunk_size + 1;

        if (threads_required <= 3 * warp_size && !sequential) {
            sequential = true;
            if (flag)
                hipMemcpyAsync(list, sorted_d, size, hipMemcpyDeviceToHost,str3);
            else
                hipMemcpyAsync(list, list_d, size, hipMemcpyDeviceToHost,str3);
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                printf("ERROR_4: %s\n", hipGetErrorString(err));
                return -1;
            }
            hipFree(list_d);
            hipFree(sorted_d);
        }
        else if (threads_required < max_threads_per_block) {
            threads_per_block = 4 * warp_size;
            dummy = threads_required / threads_per_block;
            blocks_required = (threads_required % threads_per_block == 0) ? dummy : dummy + 1;
        }
        else if (threads_required < 4 * max_active_blocks * warp_size) {
            threads_per_block = max_threads_per_block / 2;
            dummy = threads_required / threads_per_block;
            blocks_required = (threads_required % threads_per_block == 0) ? dummy : dummy + 1;
        }
        else {
            dummy = threads_required / max_active_blocks;
            // int estimated_threads_per_block = (dummy%warp_size==0) ? dummy : (dummy/warp_size + 1)*warp_size;
            int estimated_threads_per_block = (threads_required % max_active_blocks == 0) ? dummy : dummy + 1;
            if (estimated_threads_per_block > max_threads_per_block) {
                threads_per_block = max_threads_per_block;
                dummy = threads_required / max_threads_per_block;
                blocks_required = (threads_required % max_threads_per_block == 0) ? dummy : dummy + 1;
            }
            else {
                threads_per_block = estimated_threads_per_block;
                blocks_required = max_active_blocks;
            }
        }

        if (blocks_required >= max_grid_size) {
            printf("ERROR_2: Too many Blocks Required\n");
            return -1;
        }

        if (sequential) {
            double elapsed;

            START_T(elapsed);
            mergesort_gpu_seq(list, sorted, n, chunk_size);
            STOP_T(elapsed);
            
            //std::cout << "sequential elapsed: " << elapsed << "\n";
            total_elapsed_time += elapsed;
        }
        else {
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            
            //std::cout << "parallel mode\n";
            hipEventRecord(start);
            if (flag){
                mergesort_gpu_streams<<<blocks_required, threads_per_block,0,str2>>>(sorted_d, list_d, n, chunk_size);
            } else {
                mergesort_gpu_streams<<<blocks_required, threads_per_block,0,str2>>>(list_d, sorted_d, n, chunk_size);
            }

            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float elapsed;
            hipEventElapsedTime(&elapsed, start, stop);

            total_elapsed_time += elapsed;

            hipDeviceSynchronize();

           

            err = hipGetLastError();
            if (err != hipSuccess)
            {
                printf("ERROR_3: %s\n", hipGetErrorString(err));
                return -1;
            }
            flag = !flag;
        }
    }
    
    hipStreamDestroy(str1);
    hipStreamDestroy(str2);
    hipStreamDestroy(str3);

    std::cout << "merge sort time: " << total_elapsed_time << " ms\n";

    return 0;
}
