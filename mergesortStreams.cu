#include "hip/hip_runtime.h"
/** 
 * Course: High Performance Computing 2021/2022
 *
 * Lecturer: Francesco Moscato    fmoscato@unisa.it
 *
 * Group:
 * Mario Pellegrino    0622701671  m.pellegrino42@studenti.unisa.it
 * Francesco Sonnessa   0622701672   f.sonnessa@studenti.unisa.it
 *
 * Copyright (C) 2021 - All Rights Reserved 
 *
 * This file is part of Contest-CUDA.
 *
 * Contest-CUDA is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * Contest-CUDA is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with Contest-CUDA.  If not, see <http://www.gnu.org/licenses/>. 
 */

#include "main.hpp"

__device__ void merge_gpu_streams(DATATYPE *list, DATATYPE *sorted, int start, int mid, int end);
__global__ void mergesort_gpu_streams(DATATYPE *list, DATATYPE *sorted, int n, int chunk);

// // // // // // // // // // // // // // // //
//  GPU Implementation                       //
// // // // // // // // // // // // // // // //
__device__ void merge_gpu_streams(DATATYPE *list, DATATYPE *sorted, int start, int mid, int end) {
    int k = start, i = start, j = mid;
    while (i < mid || j < end)
    {
        if (j == end)
            sorted[k] = list[i++];
        else if (i == mid)
            sorted[k] = list[j++];
        else if (list[i] < list[j])
            sorted[k] = list[i++];
        else
            sorted[k] = list[j++];
        k++;
    }
}

__global__ void mergesort_gpu_streams(DATATYPE *list, DATATYPE *sorted, int n, int chunk) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int start = tid * chunk;
    if (start >= n)
        return;
    int mid, end;

    mid = min(start + chunk / 2, n);
    end = min(start + chunk, n);
    merge_gpu_streams(list, sorted, start, mid, end);
}

int mergesort_streams(DATATYPE *list, DATATYPE *sorted, int n) {

    DATATYPE *list_d;
    DATATYPE *sorted_d;
    int dummy;
    bool flag = false;
    bool sequential = false;

    int size = n * sizeof(DATATYPE);

    hipMalloc((void **)&list_d, size);
    hipMalloc((void **)&sorted_d, size);

    hipStream_t load_H2D_str, kernel_str, load_D2H_str;
    hipStreamCreate(&load_H2D_str);
    hipStreamCreate(&kernel_str);
    hipStreamCreate(&load_D2H_str);

    hipMemcpyAsync(list_d, list, size, hipMemcpyHostToDevice, load_H2D_str);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error_2: %s\n", hipGetErrorString(err));
        return -1;
    }

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    // vaues for sm_35 compute capability
    int max_active_blocks_per_sm = 16;
    if(prop.major > 3 && (prop.major < 8 && prop.minor < 5) || prop.major == 8)
        max_active_blocks_per_sm = 32;

    //const int max_active_warps_per_sm = 64;

    int warp_size = prop.warpSize;
    int max_grid_size = prop.maxGridSize[0];
    int max_threads_per_block = prop.maxThreadsPerBlock;
    int max_procs_count = prop.multiProcessorCount;

    int max_active_blocks = max_active_blocks_per_sm * max_procs_count;
    //int max_active_warps = max_active_warps_per_sm * max_procs_count;

    int chunk_size;
    float total_elapsed_time = 0;

    for (chunk_size = 2; chunk_size < 2 * n; chunk_size *= 2) {
        int blocks_required = 0, threads_per_block = 0;
        int threads_required = (n % chunk_size == 0) ? n / chunk_size : n / chunk_size + 1;

        if (threads_required <= 3 * warp_size && !sequential) {
            sequential = true;
            if (flag)
                hipMemcpyAsync(list, sorted_d, size, hipMemcpyDeviceToHost,load_D2H_str);
            else
                hipMemcpyAsync(list, list_d, size, hipMemcpyDeviceToHost,load_D2H_str);
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                printf("ERROR_4: %s\n", hipGetErrorString(err));
                return -1;
            }
            hipFree(list_d);
            hipFree(sorted_d);
        }
        else if (threads_required < max_threads_per_block) {
            threads_per_block = 4 * warp_size;
            dummy = threads_required / threads_per_block;
            blocks_required = (threads_required % threads_per_block == 0) ? dummy : dummy + 1;
        }
        else if (threads_required < 4 * max_active_blocks * warp_size) {
            threads_per_block = max_threads_per_block / 2;
            dummy = threads_required / threads_per_block;
            blocks_required = (threads_required % threads_per_block == 0) ? dummy : dummy + 1;
        }
        else {
            dummy = threads_required / max_active_blocks;
            // int estimated_threads_per_block = (dummy%warp_size==0) ? dummy : (dummy/warp_size + 1)*warp_size;
            int estimated_threads_per_block = (threads_required % max_active_blocks == 0) ? dummy : dummy + 1;
            if (estimated_threads_per_block > max_threads_per_block) {
                threads_per_block = max_threads_per_block;
                dummy = threads_required / max_threads_per_block;
                blocks_required = (threads_required % max_threads_per_block == 0) ? dummy : dummy + 1;
            }
            else {
                threads_per_block = estimated_threads_per_block;
                blocks_required = max_active_blocks;
            }
        }

        if (blocks_required >= max_grid_size) {
            printf("ERROR_2: Too many Blocks Required\n");
            return -1;
        }

        if (sequential) {
            double elapsed;

            START_T(elapsed);
            mergesort_gpu_seq(list, sorted, n, chunk_size);
            STOP_T(elapsed);
            
            //std::cout << "sequential elapsed: " << elapsed << "\n";
            total_elapsed_time += elapsed;
        }
        else {
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            
            //std::cout << "parallel mode\n";
            hipEventRecord(start);
            if (flag){
                mergesort_gpu_streams<<<blocks_required, threads_per_block,0,kernel_str>>>(sorted_d, list_d, n, chunk_size);
            } else {
                mergesort_gpu_streams<<<blocks_required, threads_per_block,0,kernel_str>>>(list_d, sorted_d, n, chunk_size);
            }

            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float elapsed;
            hipEventElapsedTime(&elapsed, start, stop);

            total_elapsed_time += elapsed;

            hipDeviceSynchronize();

           

            err = hipGetLastError();
            if (err != hipSuccess)
            {
                printf("ERROR_3: %s\n", hipGetErrorString(err));
                return -1;
            }
            flag = !flag;
        }
    }
    
    hipStreamDestroy(load_H2D_str);
    hipStreamDestroy(kernel_str);
    hipStreamDestroy(load_D2H_str);

    std::cout << "merge sort time: " << total_elapsed_time << " ms\n";

    return 0;
}
