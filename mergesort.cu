#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>

typedef int DATATYPE;

int mergesort(DATATYPE *list, DATATYPE *sorted, int n);

void merge(DATATYPE *list, DATATYPE *sorted, int start, int mid, int end);

int main(int argc, char const *argv[]) {

    struct timespec start, stop;

    int i, j;
    unsigned min_size = 2 << 16;
    unsigned max_size = 2 << 27;
    for(j=min_size; j<= max_size; j *= 2){
        std::cout << "############ LENGTH OF LIST: " << j << " ############\n";

        DATATYPE *sorted = (DATATYPE *) malloc(j*sizeof(DATATYPE));
        DATATYPE *list = (DATATYPE *) malloc(j*sizeof(DATATYPE));
        DATATYPE *list_s = (DATATYPE *) malloc(j*sizeof(DATATYPE));
        for(i=0; i<j; i++){
            list[i] = rand()%10000;
            list_s[i] = list[i];
        }
        clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
        mergesort(list, sorted, j);
        clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
        double result = (stop.tv_sec - start.tv_sec) * 1e3 + (stop.tv_nsec - start.tv_nsec) / 1e6;
        std::cout << "TIME TAKEN(Parallel GPU): "<< result << "ms\n";


        clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
        //mergesort_cpu(list_s, sorted_s, j);
        std::sort(list_s, list_s + j);
        clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
        result = (stop.tv_sec - start.tv_sec) * 1e3 + (stop.tv_nsec - start.tv_nsec) / 1e6;
        std::cout << "TIME TAKEN(Sequential CPU): "<< result << "ms\n";
        

        for(i=1; i<j; i++){
            if(sorted[i-1]>sorted[i]){
                std::cout << "WRONG ANSWER _1\n";
                return -1;
            }
        }
        std::cout << "CORRECT ANSWER\n";

        free(list);
        free(sorted);
        free(list_s);
        std::cout << "##################################################\n";
    }
    return 0;
}

// // // // // // // // // // // // // // // //
//  GPU Implementation                       //
// // // // // // // // // // // // // // // //
__device__ void merge_gpu(DATATYPE *list, DATATYPE *sorted, int start, int mid, int end)
{
    int k = start, i = start, j = mid;
    while (i < mid || j < end)
    {
        if (j == end)
            sorted[k] = list[i++];
        else if (i == mid)
            sorted[k] = list[j++];
        else if (list[i] < list[j])
            sorted[k] = list[i++];
        else
            sorted[k] = list[j++];
        k++;
    }
}

__global__ void mergesort_gpu(DATATYPE *list, DATATYPE *sorted, int n, int chunk)
{

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int start = tid * chunk;
    if (start >= n)
        return;
    int mid, end;

    mid = min(start + chunk / 2, n);
    end = min(start + chunk, n);
    merge_gpu(list, sorted, start, mid, end);
}

// Sequential Merge Sort for GPU when Number of Threads Required gets below 1 Warp Size
void mergesort_gpu_seq(DATATYPE *list, DATATYPE *sorted, int n, int chunk)
{
    int chunk_id;
    for (chunk_id = 0; chunk_id * chunk <= n; chunk_id++)
    {
        int start = chunk_id * chunk, end, mid;
        if (start >= n)
            return;
        mid = min(start + chunk / 2, n);
        end = min(start + chunk, n);
        merge(list, sorted, start, mid, end);
    }
}

void merge(DATATYPE *list, DATATYPE *sorted, int start, int mid, int end)
{
    int ti=start, i=start, j=mid;
    while (i<mid || j<end)
    {
        if (j==end) sorted[ti] = list[i++];
        else if (i==mid) sorted[ti] = list[j++];
        else if (list[i]<list[j]) sorted[ti] = list[i++];
        else sorted[ti] = list[j++];
        ti++;
    }

    for (ti=start; ti<end; ti++)
        list[ti] = sorted[ti];
}

int mergesort(DATATYPE *list, DATATYPE *sorted, int n)
{

    DATATYPE *list_d;
    DATATYPE *sorted_d;
    int dummy;
    bool flag = false;
    bool sequential = false;

    int size = n * sizeof(DATATYPE);

    hipMalloc((void **)&list_d, size);
    hipMalloc((void **)&sorted_d, size);

    hipMemcpy(list_d, list, size, hipMemcpyHostToDevice);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Error_2: %s\n", hipGetErrorString(err));
        return -1;
    }

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    //int major = prop.major;
    //int minor = prop.minor;
    /* if (major != 3 || minor != 5)
    {
        printf("The Program is Optimized only for sm_35 Compute Capability..May NOT Work for Other CCs\n");
    } */
    // vaues for sm_35 compute capability
    int max_active_blocks_per_sm = 16;
    if(prop.major > 3)
        max_active_blocks_per_sm = 32;

    //const int max_active_warps_per_sm = 64;

    int warp_size = prop.warpSize;
    int max_grid_size = prop.maxGridSize[0];
    int max_threads_per_block = prop.maxThreadsPerBlock;
    int max_procs_count = prop.multiProcessorCount;

    int max_active_blocks = max_active_blocks_per_sm * max_procs_count;
    //int max_active_warps = max_active_warps_per_sm * max_procs_count;

    int chunk_size;
    for (chunk_size = 2; chunk_size < 2 * n; chunk_size *= 2)
    {
        int blocks_required = 0, threads_per_block = 0;
        int threads_required = (n % chunk_size == 0) ? n / chunk_size : n / chunk_size + 1;

        if (threads_required <= 3 * warp_size && !sequential)
        {
            //std::cout << "sequential mode\n";
            sequential = true;
            if (flag)
                hipMemcpy(list, sorted_d, size, hipMemcpyDeviceToHost);
            else
                hipMemcpy(list, list_d, size, hipMemcpyDeviceToHost);
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                printf("ERROR_4: %s\n", hipGetErrorString(err));
                return -1;
            }
            hipFree(list_d);
            hipFree(sorted_d);
        }
        else if (threads_required < max_threads_per_block)
        {
            threads_per_block = 4 * warp_size;
            dummy = threads_required / threads_per_block;
            blocks_required = (threads_required % threads_per_block == 0) ? dummy : dummy + 1;
        }
        else if (threads_required < 4 * max_active_blocks * warp_size)
        {
            threads_per_block = max_threads_per_block / 2;
            dummy = threads_required / threads_per_block;
            blocks_required = (threads_required % threads_per_block == 0) ? dummy : dummy + 1;
        }
        else
        {
            dummy = threads_required / max_active_blocks;
            // int estimated_threads_per_block = (dummy%warp_size==0) ? dummy : (dummy/warp_size + 1)*warp_size;
            int estimated_threads_per_block = (threads_required % max_active_blocks == 0) ? dummy : dummy + 1;
            if (estimated_threads_per_block > max_threads_per_block)
            {
                threads_per_block = max_threads_per_block;
                dummy = threads_required / max_threads_per_block;
                blocks_required = (threads_required % max_threads_per_block == 0) ? dummy : dummy + 1;
            }
            else
            {
                threads_per_block = estimated_threads_per_block;
                blocks_required = max_active_blocks;
            }
        }

        if (blocks_required >= max_grid_size)
        {
            printf("ERROR_2: Too many Blocks Required\n");
            return -1;
        }

        if (sequential)
        {

            mergesort_gpu_seq(list, sorted, n, chunk_size);
        }
        else
        {
            //std::cout << "parallel mode\n";
            if (flag)
                mergesort_gpu<<<blocks_required, threads_per_block>>>(sorted_d, list_d, n, chunk_size);
            else
                mergesort_gpu<<<blocks_required, threads_per_block>>>(list_d, sorted_d, n, chunk_size);
            hipDeviceSynchronize();

            err = hipGetLastError();
            if (err != hipSuccess)
            {
                printf("ERROR_3: %s\n", hipGetErrorString(err));
                return -1;
            }
            flag = !flag;
        }
    }
    return 0;
}
